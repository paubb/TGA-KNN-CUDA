
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include <iostream>     // std::cout
#include <algorithm>    // std::sort
#include <vector>       // std::vector
#include <time.h>
using namespace std;

// Paràmetre per indicar si es vol memòria pinned (1) o no (0)
#define PINNED 0

// Paràmetre per indicar el nombre de threads per la invocació al kernel calculateDistance
#define THREADS 1024

//------------------------------------------------------------------------------------------------------------------------------------------
//FUNCIONS EXTRETES DEL CODI DELS NOSTRES COMPANYS PER FER EL MERGESORT AMB CUDA
__device__ void mergeDevice(float *list, float *sorted, float *list2, float *sorted2, int start, int mid, int end)
{
    int ti=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) {
            sorted[ti] = list[i];
            sorted2[ti] = list2[i];
            i++;
        }
        else if (i==mid) {
            sorted[ti] = list[j];
            sorted2[ti] = list2[j];
            j++;
        }
        else if (list[i]<list[j]) {
            sorted[ti] = list[i];
            sorted2[ti] = list2[i];
            i++;
        }
        else {
            sorted[ti] = list[j];
            sorted2[ti] = list2[j];
            j++;
        }
        ti++;
    }
    for (ti=start; ti<end; ti++) {
        list[ti] = sorted[ti];
        list2[ti] = sorted2[ti];
    }
}

void mergeHost(float *list, float *sorted, float *list2, float *sorted2, int start, int mid, int end)
{
    int ti=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) {
            sorted[ti] = list[i];
            sorted2[ti] = list2[i];
            i++;
        }
        else if (i==mid) {
            sorted[ti] = list[j];
            sorted2[ti] = list2[j];
            j++;
        }
        else if (list[i]<list[j]) {
            sorted[ti] = list[i];
            sorted2[ti] = list2[i];
            i++;
        }
        else {
            sorted[ti] = list[j];
            sorted2[ti] = list2[j];
            j++;
        }
        ti++;
    }
    for (ti=start; ti<end; ti++) {
        list[ti] = sorted[ti];
        list2[ti] = sorted2[ti];
    }
}

__device__ void mergeSortKernel(float *list, float *sorted, float *list2, float *sorted2, int start, int end)
{
    //Final 1: hi ha mes threads que elements del vector
    if (end-start<2)
        return;

    mergeSortKernel(list, sorted, list2, sorted2, start, start + (end-start)/2);
    mergeSortKernel(list, sorted, list2, sorted2, start + (end-start)/2, end);
    mergeDevice(list, sorted, list2, sorted2, start, start + (end-start)/2, end);
}

__global__ void callMerge(float *list, float *sorted, float *list2, float *sorted2, int chunkSize, int N) {
    if (chunkSize >= N)
        return;
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int start = tid*chunkSize;
    int end = start + chunkSize;
    if (end > N) {
        end = N;
    }
    mergeDevice(list, sorted, list2, sorted2, start, start + (end-start)/2, end);
}

__global__ void callMergeSort(float *list, float *sorted, float *list2, float *sorted2, int chunkSize, int N) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int start = tid*chunkSize;
    int end = start + chunkSize;
    if (end > N) {
        end = N;
    }
    mergeSortKernel(list, sorted, list2, sorted2, start, end);
}

void sortBlocks(float *list, float *sorted, float *list2, float *sorted2, int N) {
    int chunkSize = N;
    int start = 0;
    int end = chunkSize;
    int mid = (start+end)/2;
    
    mergeHost(list, sorted, list2, sorted2, start, mid, end);
}

//------------------------------------------------------------------------------------------------------------------------------------------
//FUNCIONS UTILITZADES EN LA VERSIÓ SEQÜENCIAL PER ORDENAR AMB QUICKSORT
// A utility function to swap two elements
void swap(float* a, float* b, float* c, float* d)
{
    int t = *a;
    int t2 = *c;
    *a = *b;
    *c = *d;
    *b = t;
    *d = t2;
}

/* This function takes last element as pivot, places
   the pivot element at its correct position in sorted
    array, and places all smaller (smaller than pivot)
   to left of pivot and all greater elements to right
   of pivot */
int partition (float *result_prediction_host, float *ref_points_host_val, int low, int high)
{
    int pivot = result_prediction_host[high];    // pivot
    int i = (low - 1);  // Index of smaller element

    for (int j = low; j <= high- 1; j++)
    {
        // If current element is smaller than or
        // equal to pivot
        if (result_prediction_host[j] <= pivot)
        {
            i++;    // increment index of smaller element
            swap(&result_prediction_host[i], &result_prediction_host[j], &ref_points_host_val[i], &ref_points_host_val[j]);
        }
    }
    swap(&result_prediction_host[i + 1], &result_prediction_host[high], &ref_points_host_val[i + 1], &ref_points_host_val[high]);
    return (i + 1);
}

/* The main function that implements QuickSort
 arr[] --> Array to be sorted,
 low  --> Starting index,
 high  --> Ending index */
void quickSort(float *result_prediction_host, float *ref_points_host_val, int low, int high)
{
    if (low < high)
    {
        /* pi is partitioning index, arr[p] is now
           at right place */
        int pi = partition(result_prediction_host, ref_points_host_val, low, high);

        // Separately sort elements before
        // partition and after partition
        quickSort(result_prediction_host, ref_points_host_val, low, pi - 1);
        quickSort(result_prediction_host, ref_points_host_val, pi + 1, high);
    }
}

//------------------------------------------------------------------------------------------------------------------------------------------
//Estructura utilitzada per representar un punt de coordenades
struct Point
{
    float x, y;     // Co-ordinate of point
};

/** Funció que es crida per fer el càlcul del nostre algorisme knn de manera seqüencial.
 * @param arr    refence points
 * @param n      number of reference points
 * @param k      number of points we want to use for the prediction
 * @param p      point we want to predict
 */
int classifyAPoint(Point arr[], int n, int k, Point p, float val[])
{
    float distances[n];

    // Fill distances of all points from p
    for (int i = 0; i < n; i++)
        distances[i] =
            sqrt((arr[i].x - p.x) * (arr[i].x - p.x) +
                 (arr[i].y - p.y) * (arr[i].y - p.y));

    // Sort the Points by distance from p
    quickSort(distances, val, 0, n-1);

    // Now consider the first k elements and only two groups.
    int freq1 = 0;     // Frequency of group 0
    int freq2 = 0;     // Frequency of group 1
    for (int i = 0; i < k; i++)
    {
        if (val[i] == 0)
            freq1++;
        else if (val[i] == 1)
            freq2++;
    }
    printf ("freq1 is %d.\n", freq1);
    printf ("freq2 is %d.\n", freq2);

    return (freq1 > freq2 ? 0 : 1);
}

//Funció per organitzar i inicialitzar els vectors del host que es necessitaran per fer càlculs al device (coordenades i valor(0,1)).
void InitHostInput(Point arr[], int n, float val[], Point p, float *ref_points_host_x, float *ref_points_host_y, float *ref_points_host_val) {

    for (int i=0; i<n; i++) {
        ref_points_host_x[i] = arr[i].x;
        ref_points_host_y[i] = arr[i].y;
        ref_points_host_val[i] = val[i];
    }

}

//Funció per inicialitzar a 0 els valors de freqüència del host.
void InitHostFreq(unsigned int *freq_host) {

    freq_host[0] = 0;
    freq_host[1] = 0;

}

//Kernel per calcular la distància euclediana entre un punt concret(p) i tots de referència.
__global__ void calculateDistance(Point p, float *ref_points_dev_x, float *ref_points_dev_y, float *result_prediction_dev) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Fill distances of all points from p

        result_prediction_dev[i] =
            sqrt((ref_points_dev_x[i] - p.x) * (ref_points_dev_x[i] - p.x) +
                 (ref_points_dev_y[i] - p.y) * (ref_points_dev_y[i] - p.y));


}

//Kernel per calcular les freqüencies dels valors 0 i 1.
__global__ void calculateFreq(float *ref_points_host_val, unsigned int *freq_dev) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;


        int j = ref_points_host_val[i];
            atomicAdd(&freq_dev[j], 1);
}

//Funció que es crida per fer el càlcul del nostre algorisme knn utilitzant CUDA i kernels.
int classifyAPointCUDA(Point arr[], float val[], int n, int k, Point p)
{   

    unsigned int numBytes;
    unsigned int nBlocks, nThreads;

    int chunkSize_sort;
    unsigned int nBytes_sort;
    unsigned int nBlocks_sort, nThreads_sort;

    float TiempoKernelDistance, TiempoSort, TiempoKernelFreq, TiempoAllOperations, TiempoProva;
    hipEvent_t E0, E1, E2, E3, E4, E5, E6, E7;

    hipEventCreate(&E0);
    hipEventCreate(&E1);
    hipEventCreate(&E2);
    hipEventCreate(&E3);
    hipEventCreate(&E4);
    hipEventCreate(&E5);
    hipEventCreate(&E6);
    hipEventCreate(&E7);

    hipEventRecord(E6, 0);

    float *ref_points_dev_x   = NULL;
    float *ref_points_dev_y   = NULL;
    float *ref_points_dev_val   = NULL;
    float *result_prediction_dev  = NULL;

    float *ref_points_host_x   = NULL;
    float *ref_points_host_y = NULL;
    float *ref_points_host_val   = NULL;
    float *result_prediction_host  = NULL;

    float *arrSorted_h, *arrSortedF_h;
    float *arrSorted_d, *arrSortedF_d;
    float *arrSorted2_h, *arrSortedF2_h;
    float *arrSorted2_d, *arrSortedF2_d;

    unsigned int *freq_dev = NULL;
    unsigned int *freq_host = NULL;


    // numero de Threads
    nThreads = THREADS;

    // numero de Blocks en cada dimension
    nBlocks = (n+nThreads-1)/nThreads;
    printf("nBlocks = %d \n", nBlocks);

    numBytes = nBlocks * nThreads * sizeof(float);
    printf("numBytes = %d \n", numBytes);

    nThreads_sort = 128;
    nBlocks_sort = 2;
    chunkSize_sort = n/(nThreads_sort*nBlocks_sort);
    nBytes_sort = n * sizeof(float);


    if (PINNED) {
        // Obtiene Memoria [pinned] en el host
        hipHostMalloc((float**)&ref_points_host_x, numBytes, hipHostMallocDefault);
        hipHostMalloc((float**)&ref_points_host_y, numBytes, hipHostMallocDefault);
        hipHostMalloc((float**)&ref_points_host_val, nBytes_sort, hipHostMallocDefault);
        hipHostMalloc((float**)&result_prediction_host, nBytes_sort, hipHostMallocDefault);

        hipHostMalloc((float**)&arrSorted_h, nBytes_sort, hipHostMallocDefault);
        hipHostMalloc((float**)&arrSortedF_h, nBytes_sort, hipHostMallocDefault);

        hipHostMalloc((float**)&freq_host, sizeof(unsigned int)*2, hipHostMallocDefault);


    } else {
        // Obtener Memoria en el host
        ref_points_host_x = (float*) malloc(numBytes);
        ref_points_host_y = (float*) malloc(numBytes);
        ref_points_host_val = (float*) malloc(nBytes_sort);
        result_prediction_host = (float*) malloc(nBytes_sort);

        arrSorted_h = (float*) malloc(nBytes_sort);
        arrSortedF_h = (float*) malloc(nBytes_sort);
        arrSorted2_h = (float*) malloc(nBytes_sort);
        arrSortedF2_h = (float*) malloc(nBytes_sort);

        freq_host = (unsigned int*) malloc(sizeof(unsigned int)*2);
    }

    InitHostInput(arr, n, val, p, ref_points_host_x, ref_points_host_y, ref_points_host_val);

    InitHostFreq(freq_host);

    // Obtener Memoria en el device
    hipMalloc((float**)&ref_points_dev_x, numBytes);
    hipMalloc((float**)&ref_points_dev_y, numBytes);
    hipMalloc((float**)&ref_points_dev_val, nBytes_sort);
    hipMalloc((float**)&result_prediction_dev, nBytes_sort);

    hipMalloc((float **) &arrSorted_d, nBytes_sort);
    hipMalloc((float **) &arrSortedF_d, nBytes_sort);
    hipMalloc((float **) &arrSorted2_d, nBytes_sort);
    hipMalloc((float **) &arrSortedF2_d, nBytes_sort);

    hipMalloc((unsigned int**)&freq_dev, sizeof(unsigned int)*2);

    // Copiar datos desde el host en el device
    hipMemcpy(ref_points_dev_x, ref_points_host_x, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(ref_points_dev_y, ref_points_host_y, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(ref_points_dev_val, ref_points_host_val, nBytes_sort, hipMemcpyHostToDevice);

    hipMemcpy(freq_dev, freq_host, sizeof(unsigned int)*2, hipMemcpyHostToDevice);


    hipEventRecord(E0, 0);

    // Ejecutar el kernel
    calculateDistance<<<nBlocks, nThreads>>>(p, ref_points_dev_x, ref_points_dev_y, result_prediction_dev);

    hipEventRecord(E1, 0); hipEventSynchronize(E1);
    hipEventElapsedTime(&TiempoKernelDistance,  E0, E1);

    // Obtener el resultado desde el host
    //cudaMemcpy(result_prediction_host, result_prediction_dev, numBytes, cudaMemcpyDeviceToHost);

    // Liberar Memoria del device
    hipFree(ref_points_dev_x);
    hipFree(ref_points_dev_y);


    hipEventRecord(E4, 0);
    // Sort the Points by distance from p

    printf("Invocació Kernel Sort <<<nBlocks, nKernels>>> (N): <<<%d, %d>>> (%d)\n", nBlocks_sort, nThreads_sort, n);

    callMergeSort<<<nBlocks_sort, nThreads_sort>>>(result_prediction_dev, arrSorted_d, ref_points_dev_val, arrSorted2_d, chunkSize_sort, n);
    int auxChunkSize = chunkSize_sort*2;
    int auxBlock = nBlocks_sort;
    int auxThread = nThreads_sort/2;

    hipFree(result_prediction_dev);
    hipFree(ref_points_dev_val);

    while (auxChunkSize < n) {
        //printf("Invocació Kernel Sort 2 <<<nBlocks, nKernels>>> (N): <<<%d, %d>>> (%d)\n", auxBlock, auxThread, n);
       callMerge<<<auxBlock, auxThread>>>(arrSorted_d, arrSortedF_d, arrSorted2_d, arrSortedF2_d, auxChunkSize, n);
       auxChunkSize = auxChunkSize*2;
       //auxThread = auxThread/2;
    }

    hipMemcpy(arrSorted_h, arrSortedF_d, nBytes_sort, hipMemcpyDeviceToHost);
    hipMemcpy(arrSorted2_h, arrSortedF2_d, nBytes_sort, hipMemcpyDeviceToHost);

    hipFree(arrSorted_d);
    hipFree(arrSortedF_d);

    sortBlocks(arrSorted_h, arrSortedF_h, arrSorted2_h, arrSortedF2_h,  n);

    //quickSort(result_prediction_host, ref_points_host_val, 0, n-1);
    /*
        for(int i = 0; i < n; i++){
        printf("L'element: %d\n", i);
        printf("La distancia: %f\n", result_prediction_host[i]);
        printf("La x: %f\n", ref_points_host_val[i]);
    }
    */

    hipEventRecord(E5, 0); hipEventSynchronize(E5);
    hipEventElapsedTime(&TiempoSort,  E4, E5);

    hipMemcpy(arrSortedF2_d, arrSortedF2_h, nBytes_sort, hipMemcpyHostToDevice);
    hipEventRecord(E2, 0);

    // Ejecutar el kernel
    calculateFreq<<<k, 1>>>(arrSortedF2_d, freq_dev);

    hipEventRecord(E3, 0); hipEventSynchronize(E3);
    hipEventElapsedTime(&TiempoKernelFreq,  E2, E3);



    TiempoAllOperations = TiempoKernelDistance + TiempoSort + TiempoKernelFreq;

    hipMemcpy(freq_host, freq_dev, sizeof(unsigned int)*2, hipMemcpyDeviceToHost);


    hipFree(ref_points_dev_val);
    hipFree(freq_dev);


    int result = -1;
    if(freq_host[0] > freq_host[1]) result = 0;
    else result = 1;

    printf ("freq1 is %d.\n", freq_host[0]);
    printf ("freq2 is %d.\n", freq_host[1]);

    printf ("The value classified to unknown point"
            " is %d.\n", result);

    printf("Invocació Kernel <<<nBlocks, nKernels>>> (N): <<<%d, %d>>> (%d)\n", nBlocks, nThreads, n);

    printf("Tiempo Kernel calculo distancia (00): %4.6f milseg\n", TiempoKernelDistance);
    printf("Tiempo Kernel calculo freq (00): %4.6f milseg\n", TiempoKernelFreq);
    printf("Tiempo Sort (00): %4.6f milseg\n", TiempoSort);
    printf("Tiempo todas las operaciones (00): %4.6f milseg\n", TiempoAllOperations);

    if (PINNED) printf("Usando Pinned Memory\n");
    else printf("NO usa Pinned Memory\n");

    if (PINNED) {
        hipHostFree(ref_points_host_x); hipHostFree(ref_points_host_y); hipHostFree(ref_points_host_val);
        hipHostFree(result_prediction_host); hipHostFree(freq_host); hipHostFree(arrSorted_h);hipHostFree(arrSortedF_h); hipHostFree(arrSorted_h);hipHostFree(arrSortedF_h);
    } else {
        free(ref_points_host_x); free(ref_points_host_y); free(ref_points_host_val); free(result_prediction_host);
        free(arrSorted_h); free(arrSortedF_h); free(freq_host); free(arrSorted2_h); free(arrSortedF2_h);
    }
    hipEventRecord(E7, 0); hipEventSynchronize(E7);
    hipEventElapsedTime(&TiempoProva,  E6, E7);

    printf("Temps total CUDA: %4.6f milseg\n", TiempoProva);

    return result;

}

//Inicialització de la k (el número de punts més propers que es vol tenir en compte) per defecte.
void InitKDefecte(int *k) {

    // Parameter to decide group of the testing point
    (*k) = 15;

}

//Inicialització del punt p (el punt sobre el que es vol fer la predicció de valor) per defecte.
void InitTestPointDefecte(struct Point *p) {

    //Test Point
    p->x = 2.5;
    p->y = 7;

}

//Inicialització conjunta de la k i la p per defecte.
void InitDefecte(int *k, struct Point *p) {
    InitKDefecte(k);
    InitTestPointDefecte(p);
}

//Funció principal del nostre programa
int main(int argc, char** argv)
{
    srand(time(0));

    //Es declaren les variables
    int n, k;
    struct Point p;

    //S'inicialitza la K, i les coordenades del Testing point
    if (argc == 1)      { InitDefecte(&k, &p); }
    else if (argc == 2) { k = atoi(argv[1]); InitTestPointDefecte(&p); }
    else if (argc == 4) { k = atoi(argv[1]); p.x = atof(argv[2]); p.y = atof(argv[3]);}
    else { printf("Usage: ./exe k TestPointCoordenadaX TestPointCoordenadaY\n"); exit(0); }

    //Es crea l'estructura sobre la qual es vol fer la predicció
    n = 131072; // Number of data points
    Point arr[n];

    float val_seq[n];
    float val_cuda[n];

    for(int i = 0; i < n; ++i) {
        arr[i].x = rand();
        arr[i].y = rand();
        val_seq[i] = rand() % 2;
        val_cuda[i] = val_seq[i];
    }

    /*for(int i = 0; i < n; i++){
        printf("x: %lf\n", arr[i].x);
        printf("y: %lf\n", arr[i].y);
        printf("val: %f\n", val[i]);
    }*/

    printf("k = %d \n", k);

    printf("The Testing Point values are:");
    printf(" x = %f", p.x);
    printf(" and");
    printf(" y = %f", p.y);
    printf("\n");

    printf("\n");
    printf("Programa Seqüencial -------------------------------------------------- \n");
    printf("\n");

    // Calculate the time taken by the sequential code: classifyAPoint function
    clock_t t;
    t = clock();
    int result = classifyAPoint(arr, n, k, p, val_seq);
    t = clock() - t;
    float time_taken = ((float)t)/(CLOCKS_PER_SEC/1000); // in mseconds

    printf ("The value classified to unknown point"
            " is %d.\n", result);

    printf ("Temps total seqüencial:"
            " %lf milseg.\n", time_taken);

    printf("\n");
    printf("Programa CUDA -------------------------------------------------------- \n");
    printf("\n");


    int result2 = classifyAPointCUDA(arr,val_cuda, n, k, p);

    printf ("The value classified to unknown point"
            " is %d.\n", result2);
}
