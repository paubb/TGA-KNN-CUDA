
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include <iostream>     // std::cout
#include <algorithm>    // std::sort
#include <vector>       // std::vector
#include <time.h>
using namespace std;

#define PINNED 1
#define THREADS 1000

struct Point
{
    int val;     // Group of point
    double x, y;     // Co-ordinate of point
    double distance; // Distance from test point
};

// Used to sort an array of points by increasing
// order of distance
bool comparison(Point a, Point b)
{
    return (a.distance < b.distance);
}

// Used to sort an array of points by increasing
// order of distance
bool comparisonNOPoints(double a, double b)
{
    return (a < b);
}

void selectionSort(double *result_prediction_host, double *ref_points_host_val, int n) {
   int i, j, min, temp, temp2;
   for (i = 0; i < n - 1; i++) {
      min = i;
      for (j = i + 1; j < n; j++)
         if (result_prediction_host[j] < result_prediction_host[min])
            min = j;
      temp = result_prediction_host[i];
      temp2 = ref_points_host_val[i];
      result_prediction_host[i] = result_prediction_host[min];
      ref_points_host_val[i] = ref_points_host_val[min];
      result_prediction_host[min] = temp;
      ref_points_host_val[min] = temp2;
   }
}

/**
 * @param arr    refence points
 * @param n      number of reference points
 * @param k      number of points we want to use for the prediction
 * @param p      point we want to predict
 */
int classifyAPoint(Point arr[], int n, int k, Point p)
{
    // Fill distances of all points from p
    for (int i = 0; i < n; i++)
        arr[i].distance =
            sqrt((arr[i].x - p.x) * (arr[i].x - p.x) +
                 (arr[i].y - p.y) * (arr[i].y - p.y));

    // Sort the Points by distance from p
    sort(arr, arr+n, comparison);

    // Now consider the first k elements and only
    // two groups
    int freq1 = 0;     // Frequency of group 0
    int freq2 = 0;     // Frequency of group 1
    for (int i = 0; i < k; i++)
    {
        if (arr[i].val == 0)
            freq1++;
        else if (arr[i].val == 1)
            freq2++;
    }
	
    printf ("freq1 is %d.\n", freq1);
    printf ("freq2 is %d.\n", freq2);

    return (freq1 > freq2 ? 0 : 1);
}

void InitHostInput(Point arr[], int n, Point p, double *ref_points_host_x, double *ref_points_host_y, double *ref_points_host_val) {

    for (int i=0; i<n; i++) {
        ref_points_host_x[i] = arr[i].x;
        ref_points_host_y[i] = arr[i].y;
        ref_points_host_val[i] = arr[i].val;
    }

}

void InitHostFreq(unsigned int *freq1_host, unsigned int *freq2_host) {

    freq1_host[0] = 0;
    freq2_host[0] = 0;

}

__global__ void calculateDistance(int n, Point p, double *ref_points_dev_x, double *ref_points_dev_y, double *result_prediction_dev) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Fill distances of all points from p

        result_prediction_dev[i] =
            sqrt((ref_points_dev_x[i] - p.x) * (ref_points_dev_x[i] - p.x) +
                 (ref_points_dev_y[i] - p.y) * (ref_points_dev_y[i] - p.y));


}

__global__ void calculateFreq(int k, double *ref_points_host_val, unsigned int *freq1_dev, unsigned int *freq2_dev) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < k) {
        if (ref_points_host_val[i] == 0) {
            atomicAdd(&freq1_dev[0], 1);
        }
        else if (ref_points_host_val[i] == 1) {
            atomicAdd(&freq2_dev[0], 1);

        }
    }
}

int classifyAPointCUDA(Point arr[], int n, int k, Point p)
{
    unsigned int N;
    unsigned int numBytes;
    unsigned int nBlocks, nThreads;

    float TiempoKernelDistance, TiempoSort, TiempoKernelFreq, TiempoAllOperations, TiempoProva;
    hipEvent_t E0, E1, E2, E3, E4, E5, E6, E7;

    hipEventCreate(&E0);
    hipEventCreate(&E1);
    hipEventCreate(&E2);
    hipEventCreate(&E3);
    hipEventCreate(&E4);
    hipEventCreate(&E5);
    hipEventCreate(&E6);
    hipEventCreate(&E7);

    hipEventRecord(E6, 0);

    double *ref_points_dev_x   = NULL;
    double *ref_points_dev_y   = NULL;
    double *ref_points_dev_val   = NULL;
    double *result_prediction_dev  = NULL;

    double *ref_points_host_x   = NULL;
    double *ref_points_host_y = NULL;
    double *ref_points_host_val   = NULL;
    double *result_prediction_host  = NULL;

    unsigned int *freq1_dev = NULL;
    unsigned int *freq2_dev = NULL;
    unsigned int *freq1_host = NULL;
    unsigned int *freq2_host = NULL;


    // numero de Threads
    nThreads = THREADS;

    // numero de Blocks en cada dimension
    nBlocks = (n+nThreads-1)/nThreads;
    printf("nBlocks = %d \n", nBlocks);

    numBytes = nBlocks * nThreads * sizeof(double);
    printf("numBytes = %d \n", numBytes);

    if (PINNED) {
        // Obtiene Memoria [pinned] en el host
        hipHostMalloc((float**)&ref_points_host_x, numBytes, hipHostMallocDefault);
        hipHostMalloc((float**)&ref_points_host_y, numBytes, hipHostMallocDefault);
        hipHostMalloc((float**)&ref_points_host_val, numBytes, hipHostMallocDefault);
        hipHostMalloc((float**)&result_prediction_host, numBytes, hipHostMallocDefault);

        hipHostMalloc((float**)&freq1_host, sizeof(unsigned int), hipHostMallocDefault);
        hipHostMalloc((float**)&freq2_host, sizeof(unsigned int), hipHostMallocDefault);

    } else {
        // Obtener Memoria en el host
        ref_points_host_x = (double*) malloc(numBytes);
        ref_points_host_y = (double*) malloc(numBytes);
        ref_points_host_val = (double*) malloc(numBytes);
        result_prediction_host = (double*) malloc(numBytes);

        freq1_host = (unsigned int*) malloc(sizeof(unsigned int));
        freq2_host = (unsigned int*) malloc(sizeof(unsigned int));
    }

    InitHostInput(arr, n, p, ref_points_host_x, ref_points_host_y, ref_points_host_val);

    InitHostFreq(freq1_host, freq2_host);

    // Obtener Memoria en el device
    hipMalloc((double**)&ref_points_dev_x, numBytes);
    hipMalloc((double**)&ref_points_dev_y, numBytes);
    hipMalloc((double**)&ref_points_dev_val, numBytes);
    hipMalloc((double**)&result_prediction_dev, numBytes);

    hipMalloc((unsigned int**)&freq1_dev, sizeof(unsigned int));
    hipMalloc((unsigned int**)&freq2_dev, sizeof(unsigned int));


    // Copiar datos desde el host en el device
    hipMemcpy(ref_points_dev_x, ref_points_host_x, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(ref_points_dev_y, ref_points_host_y, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(ref_points_dev_val, ref_points_host_val, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(result_prediction_dev, result_prediction_host,numBytes, hipMemcpyHostToDevice);

    hipMemcpy(freq1_dev, freq1_host, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(freq2_dev, freq2_host, sizeof(unsigned int), hipMemcpyHostToDevice);

    nBlocks = nBlocks-1;


    hipEventRecord(E0, 0);

    // Ejecutar el kernel
    calculateDistance<<<nBlocks, nThreads>>>(n, p, ref_points_dev_x, ref_points_dev_y, result_prediction_dev);

    hipEventRecord(E1, 0); hipEventSynchronize(E1);
    hipEventElapsedTime(&TiempoKernelDistance,  E0, E1);

    // Obtener el resultado desde el host
    hipMemcpy(result_prediction_host, result_prediction_dev, numBytes, hipMemcpyDeviceToHost);

    // Liberar Memoria del device
    hipFree(ref_points_dev_x);
    hipFree(ref_points_dev_y);
    hipFree(result_prediction_dev);

    hipEventRecord(E4, 0);
    
    // Sort the Points by distance from p
    selectionSort(result_prediction_host, ref_points_host_val, n);

    hipEventRecord(E5, 0); hipEventSynchronize(E5);
    hipEventElapsedTime(&TiempoSort,  E4, E5);

    hipEventRecord(E2, 0);
    // Ejecutar el kernel
    calculateFreq<<<k, 1>>>(k, ref_points_dev_val, freq1_dev, freq2_dev);

    hipEventRecord(E3, 0); hipEventSynchronize(E3);
    hipEventElapsedTime(&TiempoKernelFreq,  E2, E3);

    TiempoAllOperations = TiempoKernelDistance + TiempoSort + TiempoKernelFreq;

    hipMemcpy(freq1_host, freq1_dev, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(freq2_host, freq2_dev, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(ref_points_dev_val);
    hipFree(freq1_dev);
    hipFree(freq2_dev);

    int result = -1;
    if(freq1_host[0] > freq2_host[0]) result = 0;
    else result = 1;

    printf ("freq1 is %d.\n", freq1_host[0]);
    printf ("freq2 is %d.\n", freq2_host[0]);

    printf ("The value classified to unknown point"
            " is %d.\n", result);


    printf("Invocació Kernel <<<nBlocks, nKernels>>> (N): <<<%d, %d>>> (%d)\n", nBlocks, nThreads, n);

    printf("Tiempo Kernel calculo distancia (00): %4.6f milseg\n", TiempoKernelDistance);
    printf("Tiempo Kernel calculo freq (00): %4.6f milseg\n", TiempoKernelFreq);
    printf("Tiempo Sort (00): %4.6f milseg\n", TiempoSort);
    printf("Tiempo todas las operaciones (00): %4.6f milseg\n", TiempoAllOperations);

    if (PINNED) printf("Usando Pinned Memory\n");
    else printf("NO usa Pinned Memory\n");

    if (PINNED) {
        hipHostFree(ref_points_host_x); hipHostFree(ref_points_host_y); hipHostFree(ref_points_host_val);hipHostFree(result_prediction_host); hipHostFree(freq1_host); hipHostFree(freq2_host);
    } else {
        free(ref_points_host_x); free(ref_points_host_y); free(ref_points_host_val); free(result_prediction_host); free(freq1_host); free(freq2_host);
    }

    hipDeviceReset();

    hipEventRecord(E7, 0); hipEventSynchronize(E7);
    hipEventElapsedTime(&TiempoProva,  E6, E7);

    printf("Temps total CUDA: %4.6f milseg\n", TiempoProva);

    return result;
}

void InitKDefecte(int *k) {

    // Parameter to decide group of the testing point
    (*k) = 15;

}

void InitTestPointDefecte(struct Point *p) {

    //Test Point
    p->x = 2.5;
    p->y = 7;

}

void InitDefecte(int *k, struct Point *p) {
    InitKDefecte(k);
    InitTestPointDefecte(p);
}

int main(int argc, char** argv)
{
    srand(time(0));

    //Es declaren les variables
    int n, k;
    struct Point p;

    //S'inicialitza la K, i les coordenades del Testing point
    if (argc == 1)      { InitDefecte(&k, &p); }
    else if (argc == 2) { k = atoi(argv[1]); InitTestPointDefecte(&p); }
    else if (argc == 4) { k = atoi(argv[1]); p.x = atof(argv[2]); p.y = atof(argv[3]);}
    else { printf("Usage: ./exe k TestPointCoordenadaX TestPointCoordenadaY\n"); exit(0); }

    //Es crea l'estructura sobre la qual es vol fer la predicció
    n = 10000; // Number of data points
    Point arr[n];


    for(int i = 0; i < n; ++i) {
        arr[i].x = rand() % 100;
        arr[i].y = rand() % 100;
        arr[i].val = rand() % 2;
    }

    printf("k = %d \n", k);

    printf("The Testing Point values are:");
    printf(" x = %f", p.x);
    printf(" and");
    printf(" y = %f", p.y);
    printf("\n");

    printf("\n");
    printf("Programa Seqüencial -------------------------------------------------- \n");
    printf("\n");

    // Calculate the time taken by the sequential code: classifyAPoint function
    clock_t t;
    t = clock();
    int result = classifyAPoint(arr, n, k, p);
    t = clock() - t;
    float time_taken = ((float)t)/(CLOCKS_PER_SEC/1000); // in mseconds

    printf ("The value classified to unknown point"
            " is %d.\n", result);

    printf ("Temps total seqüencial:"
            " %lf milseg.\n", time_taken);

    printf("\n");
    printf("Programa CUDA -------------------------------------------------------- \n");
    printf("\n");

    int result2 = classifyAPointCUDA(arr, n, k, p);
    
	printf ("The value classified to unknown point"
            " is %d.\n", result2);
}
